#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>

#include <ATen/cuda/HIPContext.h>
#include <torch/torch.h>

#include <cstdio>
#include <stdint.h>
#include <stdexcept>
#include <limits>

#define CHECK_CUDA(x) TORCH_CHECK(x.device().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be a contiguous tensor")
#define CHECK_IS_INT(x) TORCH_CHECK(x.scalar_type() == at::ScalarType::Int, #x " must be an int tensor")
#define CHECK_IS_FLOATING(x) TORCH_CHECK(x.scalar_type() == at::ScalarType::Float || x.scalar_type() == at::ScalarType::Half || x.scalar_type() == at::ScalarType::Double, #x " must be a floating tensor")


inline constexpr __device__ float SQRT3() { return 1.7320508075688772f; }
inline constexpr __device__ float RSQRT3() { return 0.5773502691896258f; }
inline constexpr __device__ float PI() { return 3.141592653589793f; }
inline constexpr __device__ float RPI() { return 0.3183098861837907f; }


template <typename T>
inline __host__ __device__ T div_round_up(T val, T divisor) {
    return (val + divisor - 1) / divisor;
}

inline __host__ __device__ float signf(const float x) {
    return copysignf(1.0, x);
}

inline __host__ __device__ float clamp(const float x, const float min, const float max) {
    return fminf(max, fmaxf(min, x));
}

inline __host__ __device__ void swapf(float& a, float& b) {
    float c = a; a = b; b = c;
}

inline __device__ int mip_from_pos(const float x, const float y, const float z, const float max_cascade) {
    const float mx = fmaxf(fabsf(x), fmaxf(fabs(y), fabs(z)));
    int exponent;
    frexpf(mx, &exponent); // [0, 0.5) --> -1, [0.5, 1) --> 0, [1, 2) --> 1, [2, 4) --> 2, ...
    return fminf(max_cascade - 1, fmaxf(0, exponent));
}

inline __device__ int mip_from_dt(const float dt, const float H, const float max_cascade) {
    const float mx = dt * H * 0.5;
    int exponent;
    frexpf(mx, &exponent);
    return fminf(max_cascade - 1, fmaxf(0, exponent));
}

inline __host__ __device__ uint32_t __expand_bits(uint32_t v)
{
	v = (v * 0x00010001u) & 0xFF0000FFu;
	v = (v * 0x00000101u) & 0x0F00F00Fu;
	v = (v * 0x00000011u) & 0xC30C30C3u;
	v = (v * 0x00000005u) & 0x49249249u;
	return v;
}

inline __host__ __device__ uint32_t __morton3D(uint32_t x, uint32_t y, uint32_t z)
{
	uint32_t xx = __expand_bits(x);
	uint32_t yy = __expand_bits(y);
	uint32_t zz = __expand_bits(z);
	return xx | (yy << 1) | (zz << 2);
}

inline __host__ __device__ uint32_t __morton3D_invert(uint32_t x)
{
	x = x & 0x49249249;
	x = (x | (x >> 2)) & 0xc30c30c3;
	x = (x | (x >> 4)) & 0x0f00f00f;
	x = (x | (x >> 8)) & 0xff0000ff;
	x = (x | (x >> 16)) & 0x0000ffff;
	return x;
}


////////////////////////////////////////////////////
/////////////           utils          /////////////
////////////////////////////////////////////////////

// rays_o/d: [N, 3]
// nears/fars: [N]
// scalar_t should always be float in use.
template <typename scalar_t>
__global__ void kernel_near_far_from_aabb(
    const scalar_t * __restrict__ rays_o,
    const scalar_t * __restrict__ rays_d,
    const scalar_t * __restrict__ aabb,
    const uint32_t N,
    const float min_near,
    scalar_t * nears, scalar_t * fars
) {
    // parallel per ray
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= N) return;

    // locate
    rays_o += n * 3;
    rays_d += n * 3;

    const float ox = rays_o[0], oy = rays_o[1], oz = rays_o[2];
    const float dx = rays_d[0], dy = rays_d[1], dz = rays_d[2];
    const float rdx = 1 / dx, rdy = 1 / dy, rdz = 1 / dz;

    // get near far (assume cube scene)
    float near = (aabb[0] - ox) * rdx;
    float far = (aabb[3] - ox) * rdx;
    if (near > far) swapf(near, far);

    float near_y = (aabb[1] - oy) * rdy;
    float far_y = (aabb[4] - oy) * rdy;
    if (near_y > far_y) swapf(near_y, far_y);

    if (near > far_y || near_y > far) {
        nears[n] = fars[n] = std::numeric_limits<scalar_t>::max();
        return;
    }

    if (near_y > near) near = near_y;
    if (far_y < far) far = far_y;

    float near_z = (aabb[2] - oz) * rdz;
    float far_z = (aabb[5] - oz) * rdz;
    if (near_z > far_z) swapf(near_z, far_z);

    if (near > far_z || near_z > far) {
        nears[n] = fars[n] = std::numeric_limits<scalar_t>::max();
        return;
    }

    if (near_z > near) near = near_z;
    if (far_z < far) far = far_z;

    if (near < min_near) near = min_near;

    nears[n] = near;
    fars[n] = far;
}


void near_far_from_aabb(const at::Tensor rays_o, const at::Tensor rays_d, const at::Tensor aabb, const uint32_t N, const float min_near, at::Tensor nears, at::Tensor fars) {

    static constexpr uint32_t N_THREAD = 128;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    rays_o.scalar_type(), "near_far_from_aabb", ([&] {
        kernel_near_far_from_aabb<<<div_round_up(N, N_THREAD), N_THREAD>>>(rays_o.data_ptr<scalar_t>(), rays_d.data_ptr<scalar_t>(), aabb.data_ptr<scalar_t>(), N, min_near, nears.data_ptr<scalar_t>(), fars.data_ptr<scalar_t>());
    }));
}


// rays_o/d: [N, 3]
// radius: float
// coords: [N, 2]
template <typename scalar_t>
__global__ void kernel_sph_from_ray(
    const scalar_t * __restrict__ rays_o,
    const scalar_t * __restrict__ rays_d,
    const float radius,
    const uint32_t N,
    scalar_t * coords
) {
    // parallel per ray
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= N) return;

    // locate
    rays_o += n * 3;
    rays_d += n * 3;
    coords += n * 2;

    const float ox = rays_o[0], oy = rays_o[1], oz = rays_o[2];
    const float dx = rays_d[0], dy = rays_d[1], dz = rays_d[2];
    const float rdx = 1 / dx, rdy = 1 / dy, rdz = 1 / dz;

    // solve t from || o + td || = radius
    const float A = dx * dx + dy * dy + dz * dz;
    const float B = ox * dx + oy * dy + oz * dz; // in fact B / 2
    const float C = ox * ox + oy * oy + oz * oz - radius * radius;

    const float t = (- B + sqrtf(B * B - A * C)) / A; // always use the larger solution (positive)

    // solve theta, phi (assume y is the up axis)
    const float x = ox + t * dx, y = oy + t * dy, z = oz + t * dz;
    const float theta = atan2(sqrtf(x * x + z * z), y); // [0, PI)
    const float phi = atan2(z, x); // [-PI, PI)

    // normalize to [-1, 1]
    coords[0] = 2 * theta * RPI() - 1;
    coords[1] = phi * RPI();
}


void sph_from_ray(const at::Tensor rays_o, const at::Tensor rays_d, const float radius, const uint32_t N, at::Tensor coords) {

    static constexpr uint32_t N_THREAD = 128;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    rays_o.scalar_type(), "sph_from_ray", ([&] {
        kernel_sph_from_ray<<<div_round_up(N, N_THREAD), N_THREAD>>>(rays_o.data_ptr<scalar_t>(), rays_d.data_ptr<scalar_t>(), radius, N, coords.data_ptr<scalar_t>());
    }));
}


// coords: int32, [N, 3]
// indices: int32, [N]
__global__ void kernel_morton3D(
    const int * __restrict__ coords,
    const uint32_t N,
    int * indices
) {
    // parallel
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= N) return;

    // locate
    coords += n * 3;
    indices[n] = __morton3D(coords[0], coords[1], coords[2]);
}


void morton3D(const at::Tensor coords, const uint32_t N, at::Tensor indices) {
    static constexpr uint32_t N_THREAD = 128;
    kernel_morton3D<<<div_round_up(N, N_THREAD), N_THREAD>>>(coords.data_ptr<int>(), N, indices.data_ptr<int>());
}


// indices: int32, [N]
// coords: int32, [N, 3]
__global__ void kernel_morton3D_invert(
    const int * __restrict__ indices,
    const uint32_t N,
    int * coords
) {
    // parallel
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= N) return;

    // locate
    coords += n * 3;

    const int ind = indices[n];

    coords[0] = __morton3D_invert(ind >> 0);
    coords[1] = __morton3D_invert(ind >> 1);
    coords[2] = __morton3D_invert(ind >> 2);
}


void morton3D_invert(const at::Tensor indices, const uint32_t N, at::Tensor coords) {
    static constexpr uint32_t N_THREAD = 128;
    kernel_morton3D_invert<<<div_round_up(N, N_THREAD), N_THREAD>>>(indices.data_ptr<int>(), N, coords.data_ptr<int>());
}


// grid: float, [C, H, H, H]
// N: int, C * H * H * H / 8
// density_thresh: float
// bitfield: uint8, [N]
template <typename scalar_t>
__global__ void kernel_packbits(
    const scalar_t * __restrict__ grid,
    const uint32_t N,
    const float density_thresh,
    uint8_t * bitfield
) {
    // parallel per byte
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= N) return;

    // locate
    grid += n * 8;

    uint8_t bits = 0;

    #pragma unroll
    for (uint8_t i = 0; i < 8; i++) {
        bits |= (grid[i] > density_thresh) ? ((uint8_t)1 << i) : 0;
    }

    bitfield[n] = bits;
}


void packbits(const at::Tensor grid, const uint32_t N, const float density_thresh, at::Tensor bitfield) {

    static constexpr uint32_t N_THREAD = 128;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    grid.scalar_type(), "packbits", ([&] {
        kernel_packbits<<<div_round_up(N, N_THREAD), N_THREAD>>>(grid.data_ptr<scalar_t>(), N, density_thresh, bitfield.data_ptr<uint8_t>());
    }));
}


// grid: float, [C, H, H, H]
__global__ void kernel_morton3D_dilation(
    const float * __restrict__ grid,
    const uint32_t C,
    const uint32_t H,
    float * __restrict__ grid_dilation
) {
    // parallel per byte
    const uint32_t H3 = H * H * H;
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= C * H3) return;

    // locate
    const uint32_t c = n / H3;
    const uint32_t ind = n - c * H3;

    const uint32_t x = __morton3D_invert(ind >> 0);
    const uint32_t y = __morton3D_invert(ind >> 1);
    const uint32_t z = __morton3D_invert(ind >> 2);

    // manual max pool
    float res = grid[n];

    if (x + 1 < H) res = fmaxf(res, grid[c * H3 + __morton3D(x + 1, y, z)]);
    if (x > 0) res = fmaxf(res, grid[c * H3 + __morton3D(x - 1, y, z)]);
    if (y + 1 < H) res = fmaxf(res, grid[c * H3 + __morton3D(x, y + 1, z)]);
    if (y > 0) res = fmaxf(res, grid[c * H3 + __morton3D(x, y - 1, z)]);
    if (z + 1 < H) res = fmaxf(res, grid[c * H3 + __morton3D(x, y, z + 1)]);
    if (z > 0) res = fmaxf(res, grid[c * H3 + __morton3D(x, y, z - 1)]);
    
    // write
    grid_dilation[n] = res;
}

void morton3D_dilation(const at::Tensor grid, const uint32_t C, const uint32_t H, at::Tensor grid_dilation) {
    static constexpr uint32_t N_THREAD = 128;

    kernel_morton3D_dilation<<<div_round_up(C * H * H * H, N_THREAD), N_THREAD>>>(grid.data_ptr<float>(), C, H, grid_dilation.data_ptr<float>());    
}

////////////////////////////////////////////////////
/////////////         training         /////////////
////////////////////////////////////////////////////

// rays_o/d: [N, 3]
// grid: [CHHH / 8]
// xyzs, dirs, deltas: [M, 3], [M, 3], [M, 2]
// dirs: [M, 3]
// rays: [N, 3], idx, offset, num_steps
template <typename scalar_t>
__global__ void kernel_march_rays_train(
    const scalar_t * __restrict__ rays_o,
    const scalar_t * __restrict__ rays_d,  
    const uint8_t * __restrict__ grid,
    const float bound,
    const float dt_gamma, const uint32_t max_steps,
    const uint32_t N, const uint32_t C, const uint32_t H, const uint32_t M,
    const scalar_t* __restrict__ nears, 
    const scalar_t* __restrict__ fars,
    scalar_t * xyzs, scalar_t * dirs, scalar_t * deltas,
    int * rays,
    int * counter,
    const scalar_t* __restrict__ noises
) {
    // parallel per ray
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= N) return;

    // locate
    rays_o += n * 3;
    rays_d += n * 3;

    // ray marching
    const float ox = rays_o[0], oy = rays_o[1], oz = rays_o[2];
    const float dx = rays_d[0], dy = rays_d[1], dz = rays_d[2];
    const float rdx = 1 / dx, rdy = 1 / dy, rdz = 1 / dz;
    const float rH = 1 / (float)H;
    const float H3 = H * H * H;

    const float near = nears[n];
    const float far = fars[n];
    const float noise = noises[n];

    const float dt_max = 2 * SQRT3() * (1 << (C - 1)) / H;
    const float dt_min = fminf(dt_max, 2 * SQRT3() / max_steps);
    
    float t0 = near;
    
    // perturb
    t0 += clamp(t0 * dt_gamma, dt_min, dt_max) * noise;

    // first pass: estimation of num_steps
    float t = t0;
    uint32_t num_steps = 0;

    //if (t < far) printf("valid ray %d t=%f near=%f far=%f \n", n, t, near, far);
    
    while (t < far && num_steps < max_steps) {
        // current point
        const float x = clamp(ox + t * dx, -bound, bound);
        const float y = clamp(oy + t * dy, -bound, bound);
        const float z = clamp(oz + t * dz, -bound, bound);

        const float dt = clamp(t * dt_gamma, dt_min, dt_max);

        // get mip level
        const int level = max(mip_from_pos(x, y, z, C), mip_from_dt(dt, H, C)); // range in [0, C - 1]

        const float mip_bound = fminf(scalbnf(1.0f, level), bound);
        const float mip_rbound = 1 / mip_bound;
        
        // convert to nearest grid position
        const int nx = clamp(0.5 * (x * mip_rbound + 1) * H, 0.0f, (float)(H - 1));
        const int ny = clamp(0.5 * (y * mip_rbound + 1) * H, 0.0f, (float)(H - 1));
        const int nz = clamp(0.5 * (z * mip_rbound + 1) * H, 0.0f, (float)(H - 1));

        const uint32_t index = level * H3 + __morton3D(nx, ny, nz);
        const bool occ = grid[index / 8] & (1 << (index % 8));

        // if occpuied, advance a small step, and write to output
        //if (n == 0) printf("t=%f density=%f vs thresh=%f step=%d\n", t, density, density_thresh, num_steps);

        if (occ) {
            num_steps++;
            t += dt;
        // else, skip a large step (basically skip a voxel grid)
        } else {
            // calc distance to next voxel
            const float tx = (((nx + 0.5f + 0.5f * signf(dx)) * rH * 2 - 1) * mip_bound - x) * rdx;
            const float ty = (((ny + 0.5f + 0.5f * signf(dy)) * rH * 2 - 1) * mip_bound - y) * rdy;
            const float tz = (((nz + 0.5f + 0.5f * signf(dz)) * rH * 2 - 1) * mip_bound - z) * rdz;

            const float tt = t + fmaxf(0.0f, fminf(tx, fminf(ty, tz)));
            // step until next voxel
            do { 
                t += clamp(t * dt_gamma, dt_min, dt_max);
            } while (t < tt);
        }
    }

    //printf("[n=%d] num_steps=%d, near=%f, far=%f, dt=%f, max_steps=%f\n", n, num_steps, near, far, dt_min, (far - near) / dt_min);

    // second pass: really locate and write points & dirs
    uint32_t point_index = atomicAdd(counter, num_steps);
    uint32_t ray_index = atomicAdd(counter + 1, 1);
    
    //printf("[n=%d] num_steps=%d, point_index=%d, ray_index=%d\n", n, num_steps, point_index, ray_index);

    // write rays
    rays[ray_index * 3] = n;
    rays[ray_index * 3 + 1] = point_index;
    rays[ray_index * 3 + 2] = num_steps;

    if (num_steps == 0) return;
    if (point_index + num_steps > M) return;

    xyzs += point_index * 3;
    dirs += point_index * 3;
    deltas += point_index * 2;

    t = t0;
    uint32_t step = 0;

    while (t < far && step < num_steps) {
        // current point
        const float x = clamp(ox + t * dx, -bound, bound);
        const float y = clamp(oy + t * dy, -bound, bound);
        const float z = clamp(oz + t * dz, -bound, bound);

        const float dt = clamp(t * dt_gamma, dt_min, dt_max);

        // get mip level
        const int level = max(mip_from_pos(x, y, z, C), mip_from_dt(dt, H, C)); // range in [0, C - 1]

        const float mip_bound = fminf(scalbnf(1.0f, level), bound);
        const float mip_rbound = 1 / mip_bound;
        
        // convert to nearest grid position
        const int nx = clamp(0.5 * (x * mip_rbound + 1) * H, 0.0f, (float)(H - 1));
        const int ny = clamp(0.5 * (y * mip_rbound + 1) * H, 0.0f, (float)(H - 1));
        const int nz = clamp(0.5 * (z * mip_rbound + 1) * H, 0.0f, (float)(H - 1));

        // query grid
        const uint32_t index = level * H3 + __morton3D(nx, ny, nz);
        const bool occ = grid[index / 8] & (1 << (index % 8));

        // if occpuied, advance a small step, and write to output
        if (occ) {
            // write step
            xyzs[0] = x;
            xyzs[1] = y;
            xyzs[2] = z;
            dirs[0] = dx;
            dirs[1] = dy;
            dirs[2] = dz;
            t += dt;
            deltas[0] = dt;
            deltas[1] = t; // used to calc depth
            xyzs += 3;
            dirs += 3;
            deltas += 2;
            step++;
        // else, skip a large step (basically skip a voxel grid)
        } else {
            // calc distance to next voxel
            const float tx = (((nx + 0.5f + 0.5f * signf(dx)) * rH * 2 - 1) * mip_bound - x) * rdx;
            const float ty = (((ny + 0.5f + 0.5f * signf(dy)) * rH * 2 - 1) * mip_bound - y) * rdy;
            const float tz = (((nz + 0.5f + 0.5f * signf(dz)) * rH * 2 - 1) * mip_bound - z) * rdz;
            const float tt = t + fmaxf(0.0f, fminf(tx, fminf(ty, tz)));
            // step until next voxel
            do { 
                t += clamp(t * dt_gamma, dt_min, dt_max); 
            } while (t < tt);
        }
    }
}

void march_rays_train(const at::Tensor rays_o, const at::Tensor rays_d, const at::Tensor grid, const float bound, const float dt_gamma, const uint32_t max_steps, const uint32_t N, const uint32_t C, const uint32_t H, const uint32_t M, const at::Tensor nears, const at::Tensor fars, at::Tensor xyzs, at::Tensor dirs, at::Tensor deltas, at::Tensor rays, at::Tensor counter, at::Tensor noises) {

    static constexpr uint32_t N_THREAD = 128;
    
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    rays_o.scalar_type(), "march_rays_train", ([&] {
        kernel_march_rays_train<<<div_round_up(N, N_THREAD), N_THREAD>>>(rays_o.data_ptr<scalar_t>(), rays_d.data_ptr<scalar_t>(), grid.data_ptr<uint8_t>(), bound, dt_gamma, max_steps, N, C, H, M, nears.data_ptr<scalar_t>(), fars.data_ptr<scalar_t>(), xyzs.data_ptr<scalar_t>(), dirs.data_ptr<scalar_t>(), deltas.data_ptr<scalar_t>(), rays.data_ptr<int>(), counter.data_ptr<int>(), noises.data_ptr<scalar_t>());
    }));
}


// grad_xyzs/dirs: [M, 3]
// rays: [N, 3]
// deltas: [M, 2]
// grad_rays_o/d: [N, 3]
template <typename scalar_t>
__global__ void kernel_march_rays_train_backward(
    const scalar_t * __restrict__ grad_xyzs,
    const scalar_t * __restrict__ grad_dirs,  
    const int * __restrict__ rays,
    const scalar_t * __restrict__ deltas,
    const uint32_t N, const uint32_t M, 
    scalar_t * grad_rays_o,
    scalar_t * grad_rays_d
) {
    // parallel per ray
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= N) return;

    // locate
    grad_rays_o += n * 3;
    grad_rays_d += n * 3;

    uint32_t index = rays[n * 3];
    uint32_t offset = rays[n * 3 + 1];
    uint32_t num_steps = rays[n * 3 + 2];

    // empty ray, or ray that exceed max step count.
    if (num_steps == 0 || offset + num_steps > M) return;

    grad_xyzs += offset * 3;
    grad_dirs += offset * 3;
    deltas += offset * 2;

    // accumulate 
    uint32_t step = 0;
    while (step < num_steps) {

        grad_rays_o[0] += grad_xyzs[0];
        grad_rays_o[1] += grad_xyzs[1];
        grad_rays_o[2] += grad_xyzs[2];

        grad_rays_d[0] += grad_xyzs[0] * deltas[1] + grad_dirs[0];
        grad_rays_d[1] += grad_xyzs[1] * deltas[1] + grad_dirs[1];
        grad_rays_d[2] += grad_xyzs[2] * deltas[1] + grad_dirs[2];

        // locate
        grad_xyzs += 3;
        grad_dirs += 3;
        deltas += 2;

        step++;
    }
}

void march_rays_train_backward(const at::Tensor grad_xyzs, const at::Tensor grad_dirs, const at::Tensor rays, const at::Tensor deltas, const uint32_t N, const uint32_t M, at::Tensor grad_rays_o, at::Tensor grad_rays_d) {

    static constexpr uint32_t N_THREAD = 128;
    
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    grad_xyzs.scalar_type(), "march_rays_train_backward", ([&] {
        kernel_march_rays_train_backward<<<div_round_up(N, N_THREAD), N_THREAD>>>(grad_xyzs.data_ptr<scalar_t>(), grad_dirs.data_ptr<scalar_t>(), rays.data_ptr<int>(), deltas.data_ptr<scalar_t>(), N, M, grad_rays_o.data_ptr<scalar_t>(), grad_rays_d.data_ptr<scalar_t>());
    }));
}


// sigmas: [M]
// rgbs: [M, 3]
// deltas: [M, 2]
// rays: [N, 3], idx, offset, num_steps
// weights_sum: [N], final pixel alpha
// depth: [N,]
// image: [N, 3]
template <typename scalar_t>
__global__ void kernel_composite_rays_train_forward(
    const scalar_t * __restrict__ sigmas,
    const scalar_t * __restrict__ rgbs,  
    const scalar_t * __restrict__ ambient,
    const scalar_t * __restrict__ deltas,
    const int * __restrict__ rays,
    const uint32_t M, const uint32_t N, const float T_thresh, 
    scalar_t * weights_sum,
    scalar_t * ambient_sum,
    scalar_t * depth,
    scalar_t * image
) {
    // parallel per ray
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= N) return;

    // locate 
    uint32_t index = rays[n * 3];
    uint32_t offset = rays[n * 3 + 1];
    uint32_t num_steps = rays[n * 3 + 2];

    // empty ray, or ray that exceed max step count.
    if (num_steps == 0 || offset + num_steps > M) {
        weights_sum[index] = 0;
        ambient_sum[index] = 0;
        depth[index] = 0;
        image[index * 3] = 0;
        image[index * 3 + 1] = 0;
        image[index * 3 + 2] = 0;
        return;
    }

    sigmas += offset;
    rgbs += offset * 3;
    ambient += offset;
    deltas += offset * 2;

    // accumulate 
    uint32_t step = 0;

    scalar_t T = 1.0f;
    scalar_t r = 0, g = 0, b = 0, ws = 0, d = 0, amb = 0;

    while (step < num_steps) {

        const scalar_t alpha = 1.0f - __expf(- sigmas[0] * deltas[0]);
        const scalar_t weight = alpha * T;

        r += weight * rgbs[0];
        g += weight * rgbs[1];
        b += weight * rgbs[2];

        d += weight * deltas[1];
        
        ws += weight;

        amb += ambient[0];
        
        T *= 1.0f - alpha;

        // minimal remained transmittence
        if (T < T_thresh) break;

        //printf("[n=%d] num_steps=%d, alpha=%f, w=%f, T=%f, sum_dt=%f, d=%f\n", n, step, alpha, weight, T, sum_delta, d);

        // locate
        sigmas++;
        rgbs += 3;
        ambient++;
        deltas += 2;

        step++;
    }

    //printf("[n=%d] rgb=(%f, %f, %f), d=%f\n", n, r, g, b, d);

    // write
    weights_sum[index] = ws; // weights_sum
    ambient_sum[index] = amb;
    depth[index] = d;
    image[index * 3] = r;
    image[index * 3 + 1] = g;
    image[index * 3 + 2] = b;
}


void composite_rays_train_forward(const at::Tensor sigmas, const at::Tensor rgbs,  const at::Tensor ambient, const at::Tensor deltas, const at::Tensor rays, const uint32_t M, const uint32_t N, const float T_thresh, at::Tensor weights_sum, at::Tensor ambient_sum, at::Tensor depth, at::Tensor image) {

    static constexpr uint32_t N_THREAD = 128;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    sigmas.scalar_type(), "composite_rays_train_forward", ([&] {
        kernel_composite_rays_train_forward<<<div_round_up(N, N_THREAD), N_THREAD>>>(sigmas.data_ptr<scalar_t>(), rgbs.data_ptr<scalar_t>(), ambient.data_ptr<scalar_t>(), deltas.data_ptr<scalar_t>(), rays.data_ptr<int>(), M, N, T_thresh, weights_sum.data_ptr<scalar_t>(), ambient_sum.data_ptr<scalar_t>(), depth.data_ptr<scalar_t>(), image.data_ptr<scalar_t>());
    }));
}


// grad_weights_sum: [N,]
// grad: [N, 3]
// sigmas: [M]
// rgbs: [M, 3]
// deltas: [M, 2]
// rays: [N, 3], idx, offset, num_steps
// weights_sum: [N,], weights_sum here 
// image: [N, 3]
// grad_sigmas: [M]
// grad_rgbs: [M, 3]
template <typename scalar_t>
__global__ void kernel_composite_rays_train_backward(
    const scalar_t * __restrict__ grad_weights_sum,
    const scalar_t * __restrict__ grad_ambient_sum,
    const scalar_t * __restrict__ grad_image,
    const scalar_t * __restrict__ sigmas,
    const scalar_t * __restrict__ rgbs, 
    const scalar_t * __restrict__ ambient, 
    const scalar_t * __restrict__ deltas,
    const int * __restrict__ rays,
    const scalar_t * __restrict__ weights_sum,
    const scalar_t * __restrict__ ambient_sum,
    const scalar_t * __restrict__ image,
    const uint32_t M, const uint32_t N, const float T_thresh,
    scalar_t * grad_sigmas,
    scalar_t * grad_rgbs,
    scalar_t * grad_ambient
) {
    // parallel per ray
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= N) return;

    // locate 
    uint32_t index = rays[n * 3];
    uint32_t offset = rays[n * 3 + 1];
    uint32_t num_steps = rays[n * 3 + 2];

    if (num_steps == 0 || offset + num_steps > M) return;

    grad_weights_sum += index;
    grad_ambient_sum += index;
    grad_image += index * 3;
    weights_sum += index;
    ambient_sum += index;
    image += index * 3;

    sigmas += offset;
    rgbs += offset * 3;
    ambient += offset;
    deltas += offset * 2;

    grad_sigmas += offset;
    grad_rgbs += offset * 3;
    grad_ambient += offset;

    // accumulate 
    uint32_t step = 0;
    
    scalar_t T = 1.0f;
    const scalar_t r_final = image[0], g_final = image[1], b_final = image[2], ws_final = weights_sum[0];
    scalar_t r = 0, g = 0, b = 0, ws = 0;

    while (step < num_steps) {
        
        const scalar_t alpha = 1.0f - __expf(- sigmas[0] * deltas[0]);
        const scalar_t weight = alpha * T;

        r += weight * rgbs[0];
        g += weight * rgbs[1];
        b += weight * rgbs[2];
        // amb += weight * ambient[0];
        ws += weight;

        T *= 1.0f - alpha;

        // check https://note.kiui.moe/others/nerf_gradient/ for the gradient calculation.
        // write grad_rgbs
        grad_rgbs[0] = grad_image[0] * weight;
        grad_rgbs[1] = grad_image[1] * weight;
        grad_rgbs[2] = grad_image[2] * weight;
        
        // write grad_ambient
        grad_ambient[0] = grad_ambient_sum[0];

        // write grad_sigmas
        grad_sigmas[0] = deltas[0] * (
            grad_image[0] * (T * rgbs[0] - (r_final - r)) + 
            grad_image[1] * (T * rgbs[1] - (g_final - g)) + 
            grad_image[2] * (T * rgbs[2] - (b_final - b)) +
            // grad_ambient_sum[0] * (T * ambient[0] - (amb_final - amb)) +
            grad_weights_sum[0] * (1 - ws_final)
        );

        //printf("[n=%d] num_steps=%d, T=%f, grad_sigmas=%f, r_final=%f, r=%f\n", n, step, T, grad_sigmas[0], r_final, r);
        // minimal remained transmittence
        if (T < T_thresh) break;
    
        // locate
        sigmas++;
        rgbs += 3;
        // ambient++;
        deltas += 2;
        grad_sigmas++;
        grad_rgbs += 3;
        grad_ambient++;

        step++;
    }
}


void composite_rays_train_backward(const at::Tensor grad_weights_sum, const at::Tensor grad_ambient_sum, const at::Tensor grad_image, const at::Tensor sigmas, const at::Tensor rgbs, const at::Tensor ambient, const at::Tensor deltas, const at::Tensor rays, const at::Tensor weights_sum, const at::Tensor ambient_sum, const at::Tensor image, const uint32_t M, const uint32_t N, const float T_thresh, at::Tensor grad_sigmas, at::Tensor grad_rgbs, at::Tensor grad_ambient) {

    static constexpr uint32_t N_THREAD = 128;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    grad_image.scalar_type(), "composite_rays_train_backward", ([&] {
        kernel_composite_rays_train_backward<<<div_round_up(N, N_THREAD), N_THREAD>>>(grad_weights_sum.data_ptr<scalar_t>(), grad_ambient_sum.data_ptr<scalar_t>(), grad_image.data_ptr<scalar_t>(), sigmas.data_ptr<scalar_t>(), rgbs.data_ptr<scalar_t>(), ambient.data_ptr<scalar_t>(), deltas.data_ptr<scalar_t>(), rays.data_ptr<int>(), weights_sum.data_ptr<scalar_t>(), ambient_sum.data_ptr<scalar_t>(), image.data_ptr<scalar_t>(), M, N, T_thresh, grad_sigmas.data_ptr<scalar_t>(), grad_rgbs.data_ptr<scalar_t>(), grad_ambient.data_ptr<scalar_t>());
    }));
}


////////////////////////////////////////////////////
/////////////          infernce        /////////////
////////////////////////////////////////////////////

template <typename scalar_t>
__global__ void kernel_march_rays(
    const uint32_t n_alive, 
    const uint32_t n_step, 
    const int* __restrict__ rays_alive, 
    const scalar_t* __restrict__ rays_t, 
    const scalar_t* __restrict__ rays_o, 
    const scalar_t* __restrict__ rays_d, 
    const float bound,
    const float dt_gamma, const uint32_t max_steps,
    const uint32_t C, const uint32_t H,
    const uint8_t * __restrict__ grid,
    const scalar_t* __restrict__ nears,
    const scalar_t* __restrict__ fars,
    scalar_t* xyzs, scalar_t* dirs, scalar_t* deltas,
    const scalar_t* __restrict__ noises
) {
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= n_alive) return;

    const int index = rays_alive[n]; // ray id
    const float noise = noises[n];
    
    // locate
    rays_o += index * 3;
    rays_d += index * 3;
    xyzs += n * n_step * 3;
    dirs += n * n_step * 3;
    deltas += n * n_step * 2;
    
    const float ox = rays_o[0], oy = rays_o[1], oz = rays_o[2];
    const float dx = rays_d[0], dy = rays_d[1], dz = rays_d[2];
    const float rdx = 1 / dx, rdy = 1 / dy, rdz = 1 / dz;
    const float rH = 1 / (float)H;
    const float H3 = H * H * H;
    
    float t = rays_t[index]; // current ray's t
    const float near = nears[index], far = fars[index];

    const float dt_max = 2 * SQRT3() * (1 << (C - 1)) / H;
    const float dt_min = fminf(dt_max, 2 * SQRT3() / max_steps);

    // march for n_step steps, record points
    uint32_t step = 0;

    // introduce some randomness
    t += clamp(t * dt_gamma, dt_min, dt_max) * noise;

    while (t < far && step < n_step) {
        // current point
        const float x = clamp(ox + t * dx, -bound, bound);
        const float y = clamp(oy + t * dy, -bound, bound);
        const float z = clamp(oz + t * dz, -bound, bound);

        const float dt = clamp(t * dt_gamma, dt_min, dt_max);

        // get mip level
        const int level = max(mip_from_pos(x, y, z, C), mip_from_dt(dt, H, C)); // range in [0, C - 1]

        const float mip_bound = fminf(scalbnf(1, level), bound);
        const float mip_rbound = 1 / mip_bound;
        
        // convert to nearest grid position
        const int nx = clamp(0.5 * (x * mip_rbound + 1) * H, 0.0f, (float)(H - 1));
        const int ny = clamp(0.5 * (y * mip_rbound + 1) * H, 0.0f, (float)(H - 1));
        const int nz = clamp(0.5 * (z * mip_rbound + 1) * H, 0.0f, (float)(H - 1));

        const uint32_t index = level * H3 + __morton3D(nx, ny, nz);
        const bool occ = grid[index / 8] & (1 << (index % 8));

        // if occpuied, advance a small step, and write to output
        if (occ) {
            // write step
            xyzs[0] = x;
            xyzs[1] = y;
            xyzs[2] = z;
            dirs[0] = dx;
            dirs[1] = dy;
            dirs[2] = dz;
            // calc dt
            t += dt;
            deltas[0] = dt;
            deltas[1] = t; // used to calc depth
            // step
            xyzs += 3;
            dirs += 3;
            deltas += 2;
            step++;

        // else, skip a large step (basically skip a voxel grid)
        } else {
            // calc distance to next voxel
            const float tx = (((nx + 0.5f + 0.5f * signf(dx)) * rH * 2 - 1) * mip_bound - x) * rdx;
            const float ty = (((ny + 0.5f + 0.5f * signf(dy)) * rH * 2 - 1) * mip_bound - y) * rdy;
            const float tz = (((nz + 0.5f + 0.5f * signf(dz)) * rH * 2 - 1) * mip_bound - z) * rdz;
            const float tt = t + fmaxf(0.0f, fminf(tx, fminf(ty, tz)));
            // step until next voxel
            do { 
                t += clamp(t * dt_gamma, dt_min, dt_max);
            } while (t < tt);
        }
    }
}


void march_rays(const uint32_t n_alive, const uint32_t n_step, const at::Tensor rays_alive, const at::Tensor rays_t, const at::Tensor rays_o, const at::Tensor rays_d, const float bound, const float dt_gamma, const uint32_t max_steps, const uint32_t C, const uint32_t H, const at::Tensor grid, const at::Tensor near, const at::Tensor far, at::Tensor xyzs, at::Tensor dirs, at::Tensor deltas, at::Tensor noises) {
    static constexpr uint32_t N_THREAD = 128;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    rays_o.scalar_type(), "march_rays", ([&] {
        kernel_march_rays<<<div_round_up(n_alive, N_THREAD), N_THREAD>>>(n_alive, n_step, rays_alive.data_ptr<int>(), rays_t.data_ptr<scalar_t>(), rays_o.data_ptr<scalar_t>(), rays_d.data_ptr<scalar_t>(), bound, dt_gamma, max_steps, C, H, grid.data_ptr<uint8_t>(), near.data_ptr<scalar_t>(), far.data_ptr<scalar_t>(), xyzs.data_ptr<scalar_t>(), dirs.data_ptr<scalar_t>(), deltas.data_ptr<scalar_t>(), noises.data_ptr<scalar_t>());
    }));
}


template <typename scalar_t>
__global__ void kernel_composite_rays(
    const uint32_t n_alive, 
    const uint32_t n_step, 
    const float T_thresh,
    int* rays_alive, 
    scalar_t* rays_t, 
    const scalar_t* __restrict__ sigmas, 
    const scalar_t* __restrict__ rgbs, 
    const scalar_t* __restrict__ deltas, 
    scalar_t* weights_sum, scalar_t* depth, scalar_t* image
) {
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= n_alive) return;

    const int index = rays_alive[n]; // ray id
    
    // locate 
    sigmas += n * n_step;
    rgbs += n * n_step * 3;
    deltas += n * n_step * 2;
    
    rays_t += index;
    weights_sum += index;
    depth += index;
    image += index * 3;

    scalar_t t = rays_t[0]; // current ray's t
    
    scalar_t weight_sum = weights_sum[0];
    scalar_t d = depth[0];
    scalar_t r = image[0];
    scalar_t g = image[1];
    scalar_t b = image[2];

    // accumulate 
    uint32_t step = 0;
    while (step < n_step) {
        
        // ray is terminated if delta == 0
        if (deltas[0] == 0) break;
        
        const scalar_t alpha = 1.0f - __expf(- sigmas[0] * deltas[0]);

        /* 
        T_0 = 1; T_i = \prod_{j=0}^{i-1} (1 - alpha_j)
        w_i = alpha_i * T_i
        --> 
        T_i = 1 - \sum_{j=0}^{i-1} w_j
        */
        const scalar_t T = 1 - weight_sum;
        const scalar_t weight = alpha * T;
        weight_sum += weight;

        t = deltas[1];
        d += weight * t;
        r += weight * rgbs[0];
        g += weight * rgbs[1];
        b += weight * rgbs[2];

        //printf("[n=%d] num_steps=%d, alpha=%f, w=%f, T=%f, sum_dt=%f, d=%f\n", n, step, alpha, weight, T, sum_delta, d);

        // ray is terminated if T is too small
        // use a larger bound to further accelerate inference
        if (T < T_thresh) break;

        // locate
        sigmas++;
        rgbs += 3;
        deltas += 2;
        step++;
    }

    //printf("[n=%d] rgb=(%f, %f, %f), d=%f\n", n, r, g, b, d);

    // rays_alive = -1 means ray is terminated early.
    if (step < n_step) {
        rays_alive[n] = -1;
    } else {
        rays_t[0] = t;
    }

    weights_sum[0] = weight_sum; // this is the thing I needed!
    depth[0] = d;
    image[0] = r;
    image[1] = g;
    image[2] = b;
}


void composite_rays(const uint32_t n_alive, const uint32_t n_step, const float T_thresh, at::Tensor rays_alive, at::Tensor rays_t, at::Tensor sigmas, at::Tensor rgbs, at::Tensor deltas, at::Tensor weights, at::Tensor depth, at::Tensor image) {
    static constexpr uint32_t N_THREAD = 128;
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    image.scalar_type(), "composite_rays", ([&] {
        kernel_composite_rays<<<div_round_up(n_alive, N_THREAD), N_THREAD>>>(n_alive, n_step, T_thresh, rays_alive.data_ptr<int>(), rays_t.data_ptr<scalar_t>(), sigmas.data_ptr<scalar_t>(), rgbs.data_ptr<scalar_t>(), deltas.data_ptr<scalar_t>(), weights.data_ptr<scalar_t>(), depth.data_ptr<scalar_t>(), image.data_ptr<scalar_t>());
    }));
}



template <typename scalar_t>
__global__ void kernel_composite_rays_ambient(
    const uint32_t n_alive, 
    const uint32_t n_step, 
    const float T_thresh,
    int* rays_alive, 
    scalar_t* rays_t, 
    const scalar_t* __restrict__ sigmas, 
    const scalar_t* __restrict__ rgbs, 
    const scalar_t* __restrict__ deltas, 
    const scalar_t* __restrict__ ambients, 
    scalar_t* weights_sum, scalar_t* depth, scalar_t* image, scalar_t* ambient_sum
) {
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= n_alive) return;

    const int index = rays_alive[n]; // ray id
    
    // locate 
    sigmas += n * n_step;
    rgbs += n * n_step * 3;
    deltas += n * n_step * 2;
    ambients += n * n_step;
    
    rays_t += index;
    weights_sum += index;
    depth += index;
    image += index * 3;
    ambient_sum += index;

    scalar_t t = rays_t[0]; // current ray's t
    
    scalar_t weight_sum = weights_sum[0];
    scalar_t d = depth[0];
    scalar_t r = image[0];
    scalar_t g = image[1];
    scalar_t b = image[2];
    scalar_t a = ambient_sum[0];

    // accumulate 
    uint32_t step = 0;
    while (step < n_step) {
        
        // ray is terminated if delta == 0
        if (deltas[0] == 0) break;
        
        const scalar_t alpha = 1.0f - __expf(- sigmas[0] * deltas[0]);

        /* 
        T_0 = 1; T_i = \prod_{j=0}^{i-1} (1 - alpha_j)
        w_i = alpha_i * T_i
        --> 
        T_i = 1 - \sum_{j=0}^{i-1} w_j
        */
        const scalar_t T = 1 - weight_sum;
        const scalar_t weight = alpha * T;
        weight_sum += weight;

        t = deltas[1];
        d += weight * t;
        r += weight * rgbs[0];
        g += weight * rgbs[1];
        b += weight * rgbs[2];
        a += ambients[0];

        //printf("[n=%d] num_steps=%d, alpha=%f, w=%f, T=%f, sum_dt=%f, d=%f\n", n, step, alpha, weight, T, sum_delta, d);

        // ray is terminated if T is too small
        // use a larger bound to further accelerate inference
        if (T < T_thresh) break;

        // locate
        sigmas++;
        rgbs += 3;
        deltas += 2;
        step++;
        ambients++;
    }

    //printf("[n=%d] rgb=(%f, %f, %f), d=%f\n", n, r, g, b, d);

    // rays_alive = -1 means ray is terminated early.
    if (step < n_step) {
        rays_alive[n] = -1;
    } else {
        rays_t[0] = t;
    }

    weights_sum[0] = weight_sum; // this is the thing I needed!
    depth[0] = d;
    image[0] = r;
    image[1] = g;
    image[2] = b;
    ambient_sum[0] = a;
}


void composite_rays_ambient(const uint32_t n_alive, const uint32_t n_step, const float T_thresh, at::Tensor rays_alive, at::Tensor rays_t, at::Tensor sigmas, at::Tensor rgbs, at::Tensor deltas, at::Tensor ambients, at::Tensor weights, at::Tensor depth, at::Tensor image, at::Tensor ambient_sum) {
    static constexpr uint32_t N_THREAD = 128;
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    image.scalar_type(), "composite_rays_ambient", ([&] {
        kernel_composite_rays_ambient<<<div_round_up(n_alive, N_THREAD), N_THREAD>>>(n_alive, n_step, T_thresh, rays_alive.data_ptr<int>(), rays_t.data_ptr<scalar_t>(), sigmas.data_ptr<scalar_t>(), rgbs.data_ptr<scalar_t>(), deltas.data_ptr<scalar_t>(), ambients.data_ptr<scalar_t>(), weights.data_ptr<scalar_t>(), depth.data_ptr<scalar_t>(), image.data_ptr<scalar_t>(), ambient_sum.data_ptr<scalar_t>());
    }));
}






// -------------------------------- sigma ambient -----------------------------

// sigmas: [M]
// rgbs: [M, 3]
// deltas: [M, 2]
// rays: [N, 3], idx, offset, num_steps
// weights_sum: [N], final pixel alpha
// depth: [N,]
// image: [N, 3]
template <typename scalar_t>
__global__ void kernel_composite_rays_train_sigma_forward(
    const scalar_t * __restrict__ sigmas,
    const scalar_t * __restrict__ rgbs,  
    const scalar_t * __restrict__ ambient,
    const scalar_t * __restrict__ deltas,
    const int * __restrict__ rays,
    const uint32_t M, const uint32_t N, const float T_thresh, 
    scalar_t * weights_sum,
    scalar_t * ambient_sum,
    scalar_t * depth,
    scalar_t * image
) {
    // parallel per ray
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= N) return;

    // locate 
    uint32_t index = rays[n * 3];
    uint32_t offset = rays[n * 3 + 1];
    uint32_t num_steps = rays[n * 3 + 2];

    // empty ray, or ray that exceed max step count.
    if (num_steps == 0 || offset + num_steps > M) {
        weights_sum[index] = 0;
        ambient_sum[index] = 0;
        depth[index] = 0;
        image[index * 3] = 0;
        image[index * 3 + 1] = 0;
        image[index * 3 + 2] = 0;
        return;
    }

    sigmas += offset;
    rgbs += offset * 3;
    ambient += offset;
    deltas += offset * 2;

    // accumulate 
    uint32_t step = 0;

    scalar_t T = 1.0f;
    scalar_t r = 0, g = 0, b = 0, ws = 0, d = 0, amb = 0;

    while (step < num_steps) {

        const scalar_t alpha = 1.0f - __expf(- sigmas[0] * deltas[0]);
        const scalar_t weight = alpha * T;

        r += weight * rgbs[0];
        g += weight * rgbs[1];
        b += weight * rgbs[2];

        d += weight * deltas[1];
        
        ws += weight;

        amb += weight * ambient[0];
        
        T *= 1.0f - alpha;

        // minimal remained transmittence
        if (T < T_thresh) break;

        //printf("[n=%d] num_steps=%d, alpha=%f, w=%f, T=%f, sum_dt=%f, d=%f\n", n, step, alpha, weight, T, sum_delta, d);

        // locate
        sigmas++;
        rgbs += 3;
        ambient++;
        deltas += 2;

        step++;
    }

    //printf("[n=%d] rgb=(%f, %f, %f), d=%f\n", n, r, g, b, d);

    // write
    weights_sum[index] = ws; // weights_sum
    ambient_sum[index] = amb;
    depth[index] = d;
    image[index * 3] = r;
    image[index * 3 + 1] = g;
    image[index * 3 + 2] = b;
}


void composite_rays_train_sigma_forward(const at::Tensor sigmas, const at::Tensor rgbs,  const at::Tensor ambient, const at::Tensor deltas, const at::Tensor rays, const uint32_t M, const uint32_t N, const float T_thresh, at::Tensor weights_sum, at::Tensor ambient_sum, at::Tensor depth, at::Tensor image) {

    static constexpr uint32_t N_THREAD = 128;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    sigmas.scalar_type(), "composite_rays_train_sigma_forward", ([&] {
        kernel_composite_rays_train_sigma_forward<<<div_round_up(N, N_THREAD), N_THREAD>>>(sigmas.data_ptr<scalar_t>(), rgbs.data_ptr<scalar_t>(), ambient.data_ptr<scalar_t>(), deltas.data_ptr<scalar_t>(), rays.data_ptr<int>(), M, N, T_thresh, weights_sum.data_ptr<scalar_t>(), ambient_sum.data_ptr<scalar_t>(), depth.data_ptr<scalar_t>(), image.data_ptr<scalar_t>());
    }));
}


// grad_weights_sum: [N,]
// grad: [N, 3]
// sigmas: [M]
// rgbs: [M, 3]
// deltas: [M, 2]
// rays: [N, 3], idx, offset, num_steps
// weights_sum: [N,], weights_sum here 
// image: [N, 3]
// grad_sigmas: [M]
// grad_rgbs: [M, 3]
template <typename scalar_t>
__global__ void kernel_composite_rays_train_sigma_backward(
    const scalar_t * __restrict__ grad_weights_sum,
    const scalar_t * __restrict__ grad_ambient_sum,
    const scalar_t * __restrict__ grad_image,
    const scalar_t * __restrict__ sigmas,
    const scalar_t * __restrict__ rgbs, 
    const scalar_t * __restrict__ ambient, 
    const scalar_t * __restrict__ deltas,
    const int * __restrict__ rays,
    const scalar_t * __restrict__ weights_sum,
    const scalar_t * __restrict__ ambient_sum,
    const scalar_t * __restrict__ image,
    const uint32_t M, const uint32_t N, const float T_thresh,
    scalar_t * grad_sigmas,
    scalar_t * grad_rgbs,
    scalar_t * grad_ambient
) {
    // parallel per ray
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= N) return;

    // locate 
    uint32_t index = rays[n * 3];
    uint32_t offset = rays[n * 3 + 1];
    uint32_t num_steps = rays[n * 3 + 2];

    if (num_steps == 0 || offset + num_steps > M) return;

    grad_weights_sum += index;
    grad_ambient_sum += index;
    grad_image += index * 3;
    weights_sum += index;
    ambient_sum += index;
    image += index * 3;

    sigmas += offset;
    rgbs += offset * 3;
    ambient += offset;
    deltas += offset * 2;

    grad_sigmas += offset;
    grad_rgbs += offset * 3;
    grad_ambient += offset;

    // accumulate 
    uint32_t step = 0;
    
    scalar_t T = 1.0f;
    const scalar_t r_final = image[0], g_final = image[1], b_final = image[2], ws_final = weights_sum[0], amb_final = ambient_sum[0];
    scalar_t r = 0, g = 0, b = 0, ws = 0, amb = 0;

    while (step < num_steps) {
        
        const scalar_t alpha = 1.0f - __expf(- sigmas[0] * deltas[0]);
        const scalar_t weight = alpha * T;

        r += weight * rgbs[0];
        g += weight * rgbs[1];
        b += weight * rgbs[2];
        amb += weight * ambient[0];
        ws += weight;

        T *= 1.0f - alpha;

        // check https://note.kiui.moe/others/nerf_gradient/ for the gradient calculation.
        // write grad_rgbs
        grad_rgbs[0] = grad_image[0] * weight;
        grad_rgbs[1] = grad_image[1] * weight;
        grad_rgbs[2] = grad_image[2] * weight;
        
        // write grad_ambient
        grad_ambient[0] = grad_ambient_sum[0] * weight;

        // write grad_sigmas
        grad_sigmas[0] = deltas[0] * (
            grad_image[0] * (T * rgbs[0] - (r_final - r)) + 
            grad_image[1] * (T * rgbs[1] - (g_final - g)) + 
            grad_image[2] * (T * rgbs[2] - (b_final - b)) +
            grad_ambient_sum[0] * (T * ambient[0] - (amb_final - amb)) +
            grad_weights_sum[0] * (1 - ws_final)
        );

        //printf("[n=%d] num_steps=%d, T=%f, grad_sigmas=%f, r_final=%f, r=%f\n", n, step, T, grad_sigmas[0], r_final, r);
        // minimal remained transmittence
        if (T < T_thresh) break;
    
        // locate
        sigmas++;
        rgbs += 3;
        ambient++;
        deltas += 2;
        grad_sigmas++;
        grad_rgbs += 3;
        grad_ambient++;

        step++;
    }
}


void composite_rays_train_sigma_backward(const at::Tensor grad_weights_sum, const at::Tensor grad_ambient_sum, const at::Tensor grad_image, const at::Tensor sigmas, const at::Tensor rgbs, const at::Tensor ambient, const at::Tensor deltas, const at::Tensor rays, const at::Tensor weights_sum, const at::Tensor ambient_sum, const at::Tensor image, const uint32_t M, const uint32_t N, const float T_thresh, at::Tensor grad_sigmas, at::Tensor grad_rgbs, at::Tensor grad_ambient) {

    static constexpr uint32_t N_THREAD = 128;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    grad_image.scalar_type(), "composite_rays_train_sigma_backward", ([&] {
        kernel_composite_rays_train_sigma_backward<<<div_round_up(N, N_THREAD), N_THREAD>>>(grad_weights_sum.data_ptr<scalar_t>(), grad_ambient_sum.data_ptr<scalar_t>(), grad_image.data_ptr<scalar_t>(), sigmas.data_ptr<scalar_t>(), rgbs.data_ptr<scalar_t>(), ambient.data_ptr<scalar_t>(), deltas.data_ptr<scalar_t>(), rays.data_ptr<int>(), weights_sum.data_ptr<scalar_t>(), ambient_sum.data_ptr<scalar_t>(), image.data_ptr<scalar_t>(), M, N, T_thresh, grad_sigmas.data_ptr<scalar_t>(), grad_rgbs.data_ptr<scalar_t>(), grad_ambient.data_ptr<scalar_t>());
    }));
}


////////////////////////////////////////////////////
/////////////          infernce        /////////////
////////////////////////////////////////////////////


template <typename scalar_t>
__global__ void kernel_composite_rays_ambient_sigma(
    const uint32_t n_alive, 
    const uint32_t n_step, 
    const float T_thresh,
    int* rays_alive, 
    scalar_t* rays_t, 
    const scalar_t* __restrict__ sigmas, 
    const scalar_t* __restrict__ rgbs, 
    const scalar_t* __restrict__ deltas, 
    const scalar_t* __restrict__ ambients, 
    scalar_t* weights_sum, scalar_t* depth, scalar_t* image, scalar_t* ambient_sum
) {
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= n_alive) return;

    const int index = rays_alive[n]; // ray id
    
    // locate 
    sigmas += n * n_step;
    rgbs += n * n_step * 3;
    deltas += n * n_step * 2;
    ambients += n * n_step;
    
    rays_t += index;
    weights_sum += index;
    depth += index;
    image += index * 3;
    ambient_sum += index;

    scalar_t t = rays_t[0]; // current ray's t
    
    scalar_t weight_sum = weights_sum[0];
    scalar_t d = depth[0];
    scalar_t r = image[0];
    scalar_t g = image[1];
    scalar_t b = image[2];
    scalar_t a = ambient_sum[0];

    // accumulate 
    uint32_t step = 0;
    while (step < n_step) {
        
        // ray is terminated if delta == 0
        if (deltas[0] == 0) break;
        
        const scalar_t alpha = 1.0f - __expf(- sigmas[0] * deltas[0]);

        /* 
        T_0 = 1; T_i = \prod_{j=0}^{i-1} (1 - alpha_j)
        w_i = alpha_i * T_i
        --> 
        T_i = 1 - \sum_{j=0}^{i-1} w_j
        */
        const scalar_t T = 1 - weight_sum;
        const scalar_t weight = alpha * T;
        weight_sum += weight;

        t = deltas[1];
        d += weight * t;
        r += weight * rgbs[0];
        g += weight * rgbs[1];
        b += weight * rgbs[2];
        a += weight * ambients[0];

        //printf("[n=%d] num_steps=%d, alpha=%f, w=%f, T=%f, sum_dt=%f, d=%f\n", n, step, alpha, weight, T, sum_delta, d);

        // ray is terminated if T is too small
        // use a larger bound to further accelerate inference
        if (T < T_thresh) break;

        // locate
        sigmas++;
        rgbs += 3;
        deltas += 2;
        step++;
        ambients++;
    }

    //printf("[n=%d] rgb=(%f, %f, %f), d=%f\n", n, r, g, b, d);

    // rays_alive = -1 means ray is terminated early.
    if (step < n_step) {
        rays_alive[n] = -1;
    } else {
        rays_t[0] = t;
    }

    weights_sum[0] = weight_sum; // this is the thing I needed!
    depth[0] = d;
    image[0] = r;
    image[1] = g;
    image[2] = b;
    ambient_sum[0] = a;
}


void composite_rays_ambient_sigma(const uint32_t n_alive, const uint32_t n_step, const float T_thresh, at::Tensor rays_alive, at::Tensor rays_t, at::Tensor sigmas, at::Tensor rgbs, at::Tensor deltas, at::Tensor ambients, at::Tensor weights, at::Tensor depth, at::Tensor image, at::Tensor ambient_sum) {
    static constexpr uint32_t N_THREAD = 128;
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    image.scalar_type(), "composite_rays_ambient_sigma", ([&] {
        kernel_composite_rays_ambient_sigma<<<div_round_up(n_alive, N_THREAD), N_THREAD>>>(n_alive, n_step, T_thresh, rays_alive.data_ptr<int>(), rays_t.data_ptr<scalar_t>(), sigmas.data_ptr<scalar_t>(), rgbs.data_ptr<scalar_t>(), deltas.data_ptr<scalar_t>(), ambients.data_ptr<scalar_t>(), weights.data_ptr<scalar_t>(), depth.data_ptr<scalar_t>(), image.data_ptr<scalar_t>(), ambient_sum.data_ptr<scalar_t>());
    }));
}







// -------------------------------- uncertainty -----------------------------

// sigmas: [M]
// rgbs: [M, 3]
// deltas: [M, 2]
// rays: [N, 3], idx, offset, num_steps
// weights_sum: [N], final pixel alpha
// depth: [N,]
// image: [N, 3]
template <typename scalar_t>
__global__ void kernel_composite_rays_train_uncertainty_forward(
    const scalar_t * __restrict__ sigmas,
    const scalar_t * __restrict__ rgbs,  
    const scalar_t * __restrict__ ambient,
    const scalar_t * __restrict__ uncertainty,
    const scalar_t * __restrict__ deltas,
    const int * __restrict__ rays,
    const uint32_t M, const uint32_t N, const float T_thresh, 
    scalar_t * weights_sum,
    scalar_t * ambient_sum,
    scalar_t * uncertainty_sum,
    scalar_t * depth,
    scalar_t * image
) {
    // parallel per ray
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= N) return;

    // locate 
    uint32_t index = rays[n * 3];
    uint32_t offset = rays[n * 3 + 1];
    uint32_t num_steps = rays[n * 3 + 2];

    // empty ray, or ray that exceed max step count.
    if (num_steps == 0 || offset + num_steps > M) {
        weights_sum[index] = 0;
        ambient_sum[index] = 0;
        uncertainty_sum[index] = 0;
        depth[index] = 0;
        image[index * 3] = 0;
        image[index * 3 + 1] = 0;
        image[index * 3 + 2] = 0;
        return;
    }

    sigmas += offset;
    rgbs += offset * 3;
    ambient += offset;
    uncertainty += offset;
    deltas += offset * 2;

    // accumulate 
    uint32_t step = 0;

    scalar_t T = 1.0f;
    scalar_t r = 0, g = 0, b = 0, ws = 0, d = 0, amb = 0, unc = 0;

    while (step < num_steps) {

        const scalar_t alpha = 1.0f - __expf(- sigmas[0] * deltas[0]);
        const scalar_t weight = alpha * T;

        r += weight * rgbs[0];
        g += weight * rgbs[1];
        b += weight * rgbs[2];

        d += weight * deltas[1];
        
        ws += weight;

        amb += ambient[0];
        unc += weight * uncertainty[0];

        T *= 1.0f - alpha;

        // minimal remained transmittence
        if (T < T_thresh) break;

        //printf("[n=%d] num_steps=%d, alpha=%f, w=%f, T=%f, sum_dt=%f, d=%f\n", n, step, alpha, weight, T, sum_delta, d);

        // locate
        sigmas++;
        rgbs += 3;
        ambient++;
        uncertainty++;
        deltas += 2;

        step++;
    }

    //printf("[n=%d] rgb=(%f, %f, %f), d=%f\n", n, r, g, b, d);

    // write
    weights_sum[index] = ws; // weights_sum
    ambient_sum[index] = amb;
    uncertainty_sum[index] = unc;
    depth[index] = d;
    image[index * 3] = r;
    image[index * 3 + 1] = g;
    image[index * 3 + 2] = b;
}


void composite_rays_train_uncertainty_forward(const at::Tensor sigmas, const at::Tensor rgbs, const at::Tensor ambient, const at::Tensor uncertainty, const at::Tensor deltas, const at::Tensor rays, const uint32_t M, const uint32_t N, const float T_thresh, at::Tensor weights_sum, at::Tensor ambient_sum, at::Tensor uncertainty_sum, at::Tensor depth, at::Tensor image) {

    static constexpr uint32_t N_THREAD = 128;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    sigmas.scalar_type(), "composite_rays_train_uncertainty_forward", ([&] {
        kernel_composite_rays_train_uncertainty_forward<<<div_round_up(N, N_THREAD), N_THREAD>>>(sigmas.data_ptr<scalar_t>(), rgbs.data_ptr<scalar_t>(), ambient.data_ptr<scalar_t>(), uncertainty.data_ptr<scalar_t>(), deltas.data_ptr<scalar_t>(), rays.data_ptr<int>(), M, N, T_thresh, weights_sum.data_ptr<scalar_t>(), ambient_sum.data_ptr<scalar_t>(), uncertainty_sum.data_ptr<scalar_t>(), depth.data_ptr<scalar_t>(), image.data_ptr<scalar_t>());
    }));
}


// grad_weights_sum: [N,]
// grad: [N, 3]
// sigmas: [M]
// rgbs: [M, 3]
// deltas: [M, 2]
// rays: [N, 3], idx, offset, num_steps
// weights_sum: [N,], weights_sum here 
// image: [N, 3]
// grad_sigmas: [M]
// grad_rgbs: [M, 3]
template <typename scalar_t>
__global__ void kernel_composite_rays_train_uncertainty_backward(
    const scalar_t * __restrict__ grad_weights_sum,
    const scalar_t * __restrict__ grad_ambient_sum,
    const scalar_t * __restrict__ grad_uncertainty_sum,
    const scalar_t * __restrict__ grad_image,
    const scalar_t * __restrict__ sigmas,
    const scalar_t * __restrict__ rgbs, 
    const scalar_t * __restrict__ ambient, 
    const scalar_t * __restrict__ uncertainty, 
    const scalar_t * __restrict__ deltas,
    const int * __restrict__ rays,
    const scalar_t * __restrict__ weights_sum,
    const scalar_t * __restrict__ ambient_sum,
    const scalar_t * __restrict__ uncertainty_sum,
    const scalar_t * __restrict__ image,
    const uint32_t M, const uint32_t N, const float T_thresh,
    scalar_t * grad_sigmas,
    scalar_t * grad_rgbs,
    scalar_t * grad_ambient,
    scalar_t * grad_uncertainty
) {
    // parallel per ray
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= N) return;

    // locate 
    uint32_t index = rays[n * 3];
    uint32_t offset = rays[n * 3 + 1];
    uint32_t num_steps = rays[n * 3 + 2];

    if (num_steps == 0 || offset + num_steps > M) return;

    grad_weights_sum += index;
    grad_ambient_sum += index;
    grad_uncertainty_sum += index;
    grad_image += index * 3;
    weights_sum += index;
    ambient_sum += index;
    uncertainty_sum += index;
    image += index * 3;

    sigmas += offset;
    rgbs += offset * 3;
    ambient += offset;
    uncertainty += offset;
    deltas += offset * 2;

    grad_sigmas += offset;
    grad_rgbs += offset * 3;
    grad_ambient += offset;
    grad_uncertainty += offset;

    // accumulate 
    uint32_t step = 0;
    
    scalar_t T = 1.0f;
    const scalar_t r_final = image[0], g_final = image[1], b_final = image[2], ws_final = weights_sum[0], amb_final = ambient_sum[0], unc_final = uncertainty_sum[0];
    scalar_t r = 0, g = 0, b = 0, ws = 0, amb = 0, unc = 0;

    while (step < num_steps) {
        
        const scalar_t alpha = 1.0f - __expf(- sigmas[0] * deltas[0]);
        const scalar_t weight = alpha * T;

        r += weight * rgbs[0];
        g += weight * rgbs[1];
        b += weight * rgbs[2];
        // amb += ambient[0];
        unc += weight * uncertainty[0];
        ws += weight;

        T *= 1.0f - alpha;

        // check https://note.kiui.moe/others/nerf_gradient/ for the gradient calculation.
        // write grad_rgbs
        grad_rgbs[0] = grad_image[0] * weight;
        grad_rgbs[1] = grad_image[1] * weight;
        grad_rgbs[2] = grad_image[2] * weight;
        
        // write grad_ambient
        grad_ambient[0] = grad_ambient_sum[0];

        // write grad_unc
        grad_uncertainty[0] = grad_uncertainty_sum[0] * weight;

        // write grad_sigmas
        grad_sigmas[0] = deltas[0] * (
            grad_image[0] * (T * rgbs[0] - (r_final - r)) + 
            grad_image[1] * (T * rgbs[1] - (g_final - g)) + 
            grad_image[2] * (T * rgbs[2] - (b_final - b)) +
            // grad_ambient_sum[0] * (T * ambient[0] - (amb_final - amb)) +
            grad_uncertainty_sum[0] * (T * uncertainty[0] - (unc_final - unc)) +
            grad_weights_sum[0] * (1 - ws_final)
        );

        //printf("[n=%d] num_steps=%d, T=%f, grad_sigmas=%f, r_final=%f, r=%f\n", n, step, T, grad_sigmas[0], r_final, r);
        // minimal remained transmittence
        if (T < T_thresh) break;
    
        // locate
        sigmas++;
        rgbs += 3;
        // ambient++;
        uncertainty++;
        deltas += 2;
        grad_sigmas++;
        grad_rgbs += 3;
        grad_ambient++;
        grad_uncertainty++;

        step++;
    }
}


void composite_rays_train_uncertainty_backward(const at::Tensor grad_weights_sum, const at::Tensor grad_ambient_sum, const at::Tensor grad_uncertainty_sum, const at::Tensor grad_image, const at::Tensor sigmas, const at::Tensor rgbs, const at::Tensor ambient, const at::Tensor uncertainty, const at::Tensor deltas, const at::Tensor rays, const at::Tensor weights_sum, const at::Tensor ambient_sum, const at::Tensor uncertainty_sum, const at::Tensor image, const uint32_t M, const uint32_t N, const float T_thresh, at::Tensor grad_sigmas, at::Tensor grad_rgbs, at::Tensor grad_ambient, at::Tensor grad_uncertainty) {

    static constexpr uint32_t N_THREAD = 128;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    grad_image.scalar_type(), "composite_rays_train_uncertainty_backward", ([&] {
        kernel_composite_rays_train_uncertainty_backward<<<div_round_up(N, N_THREAD), N_THREAD>>>(grad_weights_sum.data_ptr<scalar_t>(), grad_ambient_sum.data_ptr<scalar_t>(), grad_uncertainty_sum.data_ptr<scalar_t>(), grad_image.data_ptr<scalar_t>(), sigmas.data_ptr<scalar_t>(), rgbs.data_ptr<scalar_t>(), ambient.data_ptr<scalar_t>(), uncertainty.data_ptr<scalar_t>(), deltas.data_ptr<scalar_t>(), rays.data_ptr<int>(), weights_sum.data_ptr<scalar_t>(), ambient_sum.data_ptr<scalar_t>(), uncertainty_sum.data_ptr<scalar_t>(), image.data_ptr<scalar_t>(), M, N, T_thresh, grad_sigmas.data_ptr<scalar_t>(), grad_rgbs.data_ptr<scalar_t>(), grad_ambient.data_ptr<scalar_t>(), grad_uncertainty.data_ptr<scalar_t>());
    }));
}


////////////////////////////////////////////////////
/////////////          infernce        /////////////
////////////////////////////////////////////////////


template <typename scalar_t>
__global__ void kernel_composite_rays_uncertainty(
    const uint32_t n_alive, 
    const uint32_t n_step, 
    const float T_thresh,
    int* rays_alive, 
    scalar_t* rays_t, 
    const scalar_t* __restrict__ sigmas, 
    const scalar_t* __restrict__ rgbs, 
    const scalar_t* __restrict__ deltas, 
    const scalar_t* __restrict__ ambients, 
    const scalar_t* __restrict__ uncertainties, 
    scalar_t* weights_sum, scalar_t* depth, scalar_t* image, scalar_t* ambient_sum, scalar_t* uncertainty_sum
) {
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= n_alive) return;

    const int index = rays_alive[n]; // ray id
    
    // locate 
    sigmas += n * n_step;
    rgbs += n * n_step * 3;
    deltas += n * n_step * 2;
    ambients += n * n_step;
    uncertainties += n * n_step;

    rays_t += index;
    weights_sum += index;
    depth += index;
    image += index * 3;
    ambient_sum += index;
    uncertainty_sum += index;

    scalar_t t = rays_t[0]; // current ray's t
    
    scalar_t weight_sum = weights_sum[0];
    scalar_t d = depth[0];
    scalar_t r = image[0];
    scalar_t g = image[1];
    scalar_t b = image[2];
    scalar_t a = ambient_sum[0];
    scalar_t u = uncertainty_sum[0];

    // accumulate 
    uint32_t step = 0;
    while (step < n_step) {
        
        // ray is terminated if delta == 0
        if (deltas[0] == 0) break;
        
        const scalar_t alpha = 1.0f - __expf(- sigmas[0] * deltas[0]);

        /* 
        T_0 = 1; T_i = \prod_{j=0}^{i-1} (1 - alpha_j)
        w_i = alpha_i * T_i
        --> 
        T_i = 1 - \sum_{j=0}^{i-1} w_j
        */
        const scalar_t T = 1 - weight_sum;
        const scalar_t weight = alpha * T;
        weight_sum += weight;

        t = deltas[1];
        d += weight * t;
        r += weight * rgbs[0];
        g += weight * rgbs[1];
        b += weight * rgbs[2];
        a += ambients[0];
        u += weight * uncertainties[0];

        //printf("[n=%d] num_steps=%d, alpha=%f, w=%f, T=%f, sum_dt=%f, d=%f\n", n, step, alpha, weight, T, sum_delta, d);

        // ray is terminated if T is too small
        // use a larger bound to further accelerate inference
        if (T < T_thresh) break;

        // locate
        sigmas++;
        rgbs += 3;
        deltas += 2;
        step++;
        ambients++;
        uncertainties++;
    }

    //printf("[n=%d] rgb=(%f, %f, %f), d=%f\n", n, r, g, b, d);

    // rays_alive = -1 means ray is terminated early.
    if (step < n_step) {
        rays_alive[n] = -1;
    } else {
        rays_t[0] = t;
    }

    weights_sum[0] = weight_sum; // this is the thing I needed!
    depth[0] = d;
    image[0] = r;
    image[1] = g;
    image[2] = b;
    ambient_sum[0] = a;
    uncertainty_sum[0] = u;
}


void composite_rays_uncertainty(const uint32_t n_alive, const uint32_t n_step, const float T_thresh, at::Tensor rays_alive, at::Tensor rays_t, at::Tensor sigmas, at::Tensor rgbs, at::Tensor deltas, at::Tensor ambients, at::Tensor uncertainties, at::Tensor weights, at::Tensor depth, at::Tensor image, at::Tensor ambient_sum, at::Tensor uncertainty_sum) {
    static constexpr uint32_t N_THREAD = 128;
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    image.scalar_type(), "composite_rays_uncertainty", ([&] {
        kernel_composite_rays_uncertainty<<<div_round_up(n_alive, N_THREAD), N_THREAD>>>(n_alive, n_step, T_thresh, rays_alive.data_ptr<int>(), rays_t.data_ptr<scalar_t>(), sigmas.data_ptr<scalar_t>(), rgbs.data_ptr<scalar_t>(), deltas.data_ptr<scalar_t>(), ambients.data_ptr<scalar_t>(), uncertainties.data_ptr<scalar_t>(), weights.data_ptr<scalar_t>(), depth.data_ptr<scalar_t>(), image.data_ptr<scalar_t>(), ambient_sum.data_ptr<scalar_t>(), uncertainty_sum.data_ptr<scalar_t>());
    }));
}




// -------------------------------- triplane -----------------------------

// sigmas: [M]
// rgbs: [M, 3]
// deltas: [M, 2]
// rays: [N, 3], idx, offset, num_steps
// weights_sum: [N], final pixel alpha
// depth: [N,]
// image: [N, 3]
template <typename scalar_t>
__global__ void kernel_composite_rays_train_triplane_forward(
    const scalar_t * __restrict__ sigmas,
    const scalar_t * __restrict__ rgbs,  
    const scalar_t * __restrict__ amb_aud,
    const scalar_t * __restrict__ amb_eye,
    const scalar_t * __restrict__ uncertainty,
    const scalar_t * __restrict__ deltas,
    const int * __restrict__ rays,
    const uint32_t M, const uint32_t N, const float T_thresh, 
    scalar_t * weights_sum,
    scalar_t * amb_aud_sum,
    scalar_t * amb_eye_sum,
    scalar_t * uncertainty_sum,
    scalar_t * depth,
    scalar_t * image
) {
    // parallel per ray
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= N) return;

    // locate 
    uint32_t index = rays[n * 3];
    uint32_t offset = rays[n * 3 + 1];
    uint32_t num_steps = rays[n * 3 + 2];

    // empty ray, or ray that exceed max step count.
    if (num_steps == 0 || offset + num_steps > M) {
        weights_sum[index] = 0;
        amb_aud_sum[index] = 0;
        amb_eye_sum[index] = 0;
        uncertainty_sum[index] = 0;
        depth[index] = 0;
        image[index * 3] = 0;
        image[index * 3 + 1] = 0;
        image[index * 3 + 2] = 0;
        return;
    }

    sigmas += offset;
    rgbs += offset * 3;
    amb_aud += offset;
    amb_eye += offset;
    uncertainty += offset;
    deltas += offset * 2;

    // accumulate 
    uint32_t step = 0;

    scalar_t T = 1.0f;
    scalar_t r = 0, g = 0, b = 0, ws = 0, d = 0, a_aud = 0, a_eye=0, unc = 0;

    while (step < num_steps) {

        const scalar_t alpha = 1.0f - __expf(- sigmas[0] * deltas[0]);
        const scalar_t weight = alpha * T;

        r += weight * rgbs[0];
        g += weight * rgbs[1];
        b += weight * rgbs[2];

        d += weight * deltas[1];
        
        ws += weight;

        a_aud += amb_aud[0];
        a_eye += amb_eye[0];
        unc += weight * uncertainty[0];

        T *= 1.0f - alpha;

        // minimal remained transmittence
        if (T < T_thresh) break;

        //printf("[n=%d] num_steps=%d, alpha=%f, w=%f, T=%f, sum_dt=%f, d=%f\n", n, step, alpha, weight, T, sum_delta, d);

        // locate
        sigmas++;
        rgbs += 3;
        amb_aud++;
        amb_eye++;
        uncertainty++;
        deltas += 2;

        step++;
    }

    //printf("[n=%d] rgb=(%f, %f, %f), d=%f\n", n, r, g, b, d);

    // write
    weights_sum[index] = ws; // weights_sum
    amb_aud_sum[index] = a_aud;
    amb_eye_sum[index] = a_eye;
    uncertainty_sum[index] = unc;
    depth[index] = d;
    image[index * 3] = r;
    image[index * 3 + 1] = g;
    image[index * 3 + 2] = b;
}


void composite_rays_train_triplane_forward(const at::Tensor sigmas, const at::Tensor rgbs, const at::Tensor amb_aud, const at::Tensor amb_eye, const at::Tensor uncertainty, const at::Tensor deltas, const at::Tensor rays, const uint32_t M, const uint32_t N, const float T_thresh, at::Tensor weights_sum, at::Tensor amb_aud_sum, at::Tensor amb_eye_sum, at::Tensor uncertainty_sum, at::Tensor depth, at::Tensor image) {

    static constexpr uint32_t N_THREAD = 128;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    sigmas.scalar_type(), "composite_rays_train_triplane_forward", ([&] {
        kernel_composite_rays_train_triplane_forward<<<div_round_up(N, N_THREAD), N_THREAD>>>(sigmas.data_ptr<scalar_t>(), rgbs.data_ptr<scalar_t>(), amb_aud.data_ptr<scalar_t>(), amb_eye.data_ptr<scalar_t>(), uncertainty.data_ptr<scalar_t>(), deltas.data_ptr<scalar_t>(), rays.data_ptr<int>(), M, N, T_thresh, weights_sum.data_ptr<scalar_t>(), amb_aud_sum.data_ptr<scalar_t>(), amb_eye_sum.data_ptr<scalar_t>(), uncertainty_sum.data_ptr<scalar_t>(), depth.data_ptr<scalar_t>(), image.data_ptr<scalar_t>());
    }));
}


// grad_weights_sum: [N,]
// grad: [N, 3]
// sigmas: [M]
// rgbs: [M, 3]
// deltas: [M, 2]
// rays: [N, 3], idx, offset, num_steps
// weights_sum: [N,], weights_sum here 
// image: [N, 3]
// grad_sigmas: [M]
// grad_rgbs: [M, 3]
template <typename scalar_t>
__global__ void kernel_composite_rays_train_triplane_backward(
    const scalar_t * __restrict__ grad_weights_sum,
    const scalar_t * __restrict__ grad_amb_aud_sum,
    const scalar_t * __restrict__ grad_amb_eye_sum,
    const scalar_t * __restrict__ grad_uncertainty_sum,
    const scalar_t * __restrict__ grad_image,
    const scalar_t * __restrict__ sigmas,
    const scalar_t * __restrict__ rgbs, 
    const scalar_t * __restrict__ amb_aud, 
    const scalar_t * __restrict__ amb_eye, 
    const scalar_t * __restrict__ uncertainty, 
    const scalar_t * __restrict__ deltas,
    const int * __restrict__ rays,
    const scalar_t * __restrict__ weights_sum,
    const scalar_t * __restrict__ amb_aud_sum,
    const scalar_t * __restrict__ amb_eye_sum,
    const scalar_t * __restrict__ uncertainty_sum,
    const scalar_t * __restrict__ image,
    const uint32_t M, const uint32_t N, const float T_thresh,
    scalar_t * grad_sigmas,
    scalar_t * grad_rgbs,
    scalar_t * grad_amb_aud,
    scalar_t * grad_amb_eye,
    scalar_t * grad_uncertainty
) {
    // parallel per ray
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= N) return;

    // locate 
    uint32_t index = rays[n * 3];
    uint32_t offset = rays[n * 3 + 1];
    uint32_t num_steps = rays[n * 3 + 2];

    if (num_steps == 0 || offset + num_steps > M) return;

    grad_weights_sum += index;
    grad_amb_aud_sum += index;
    grad_amb_eye_sum += index;
    grad_uncertainty_sum += index;
    grad_image += index * 3;
    weights_sum += index;
    amb_aud_sum += index;
    amb_eye_sum += index;
    uncertainty_sum += index;
    image += index * 3;

    sigmas += offset;
    rgbs += offset * 3;
    amb_aud += offset;
    amb_eye += offset;
    uncertainty += offset;
    deltas += offset * 2;

    grad_sigmas += offset;
    grad_rgbs += offset * 3;
    grad_amb_aud += offset;
    grad_amb_eye += offset;
    grad_uncertainty += offset;

    // accumulate 
    uint32_t step = 0;
    
    scalar_t T = 1.0f;
    const scalar_t r_final = image[0], g_final = image[1], b_final = image[2], ws_final = weights_sum[0], unc_final = uncertainty_sum[0];
    scalar_t r = 0, g = 0, b = 0, ws = 0, amb = 0, unc = 0;

    while (step < num_steps) {
        
        const scalar_t alpha = 1.0f - __expf(- sigmas[0] * deltas[0]);
        const scalar_t weight = alpha * T;

        r += weight * rgbs[0];
        g += weight * rgbs[1];
        b += weight * rgbs[2];
        // amb += ambient[0];
        unc += weight * uncertainty[0];
        ws += weight;

        T *= 1.0f - alpha;

        // check https://note.kiui.moe/others/nerf_gradient/ for the gradient calculation.
        // write grad_rgbs
        grad_rgbs[0] = grad_image[0] * weight;
        grad_rgbs[1] = grad_image[1] * weight;
        grad_rgbs[2] = grad_image[2] * weight;
        
        // write grad_ambient
        grad_amb_aud[0] = grad_amb_aud_sum[0];
        grad_amb_eye[0] = grad_amb_eye_sum[0];

        // write grad_unc
        grad_uncertainty[0] = grad_uncertainty_sum[0] * weight;

        // write grad_sigmas
        grad_sigmas[0] = deltas[0] * (
            grad_image[0] * (T * rgbs[0] - (r_final - r)) + 
            grad_image[1] * (T * rgbs[1] - (g_final - g)) + 
            grad_image[2] * (T * rgbs[2] - (b_final - b)) +
            // grad_ambient_sum[0] * (T * ambient[0] - (amb_final - amb)) +
            grad_uncertainty_sum[0] * (T * uncertainty[0] - (unc_final - unc)) +
            grad_weights_sum[0] * (1 - ws_final)
        );

        //printf("[n=%d] num_steps=%d, T=%f, grad_sigmas=%f, r_final=%f, r=%f\n", n, step, T, grad_sigmas[0], r_final, r);
        // minimal remained transmittence
        if (T < T_thresh) break;
    
        // locate
        sigmas++;
        rgbs += 3;
        // ambient++;
        uncertainty++;
        deltas += 2;
        grad_sigmas++;
        grad_rgbs += 3;
        grad_amb_aud++;
        grad_amb_eye++;
        grad_uncertainty++;

        step++;
    }
}


void composite_rays_train_triplane_backward(const at::Tensor grad_weights_sum, const at::Tensor grad_amb_aud_sum, const at::Tensor grad_amb_eye_sum, const at::Tensor grad_uncertainty_sum, const at::Tensor grad_image, const at::Tensor sigmas, const at::Tensor rgbs, const at::Tensor amb_aud, const at::Tensor amb_eye, const at::Tensor uncertainty, const at::Tensor deltas, const at::Tensor rays, const at::Tensor weights_sum, const at::Tensor amb_aud_sum, const at::Tensor amb_eye_sum, const at::Tensor uncertainty_sum, const at::Tensor image, const uint32_t M, const uint32_t N, const float T_thresh, at::Tensor grad_sigmas, at::Tensor grad_rgbs, at::Tensor grad_amb_aud, at::Tensor grad_amb_eye, at::Tensor grad_uncertainty) {

    static constexpr uint32_t N_THREAD = 128;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    grad_image.scalar_type(), "composite_rays_train_triplane_backward", ([&] {
        kernel_composite_rays_train_triplane_backward<<<div_round_up(N, N_THREAD), N_THREAD>>>(grad_weights_sum.data_ptr<scalar_t>(), grad_amb_aud_sum.data_ptr<scalar_t>(), grad_amb_eye_sum.data_ptr<scalar_t>(), grad_uncertainty_sum.data_ptr<scalar_t>(), grad_image.data_ptr<scalar_t>(), sigmas.data_ptr<scalar_t>(), rgbs.data_ptr<scalar_t>(), amb_aud.data_ptr<scalar_t>(), amb_eye.data_ptr<scalar_t>(), uncertainty.data_ptr<scalar_t>(), deltas.data_ptr<scalar_t>(), rays.data_ptr<int>(), weights_sum.data_ptr<scalar_t>(), amb_aud_sum.data_ptr<scalar_t>(), amb_eye_sum.data_ptr<scalar_t>(), uncertainty_sum.data_ptr<scalar_t>(), image.data_ptr<scalar_t>(), M, N, T_thresh, grad_sigmas.data_ptr<scalar_t>(), grad_rgbs.data_ptr<scalar_t>(), grad_amb_aud.data_ptr<scalar_t>(), grad_amb_eye.data_ptr<scalar_t>(), grad_uncertainty.data_ptr<scalar_t>());
    }));
}


////////////////////////////////////////////////////
/////////////          infernce        /////////////
////////////////////////////////////////////////////


template <typename scalar_t>
__global__ void kernel_composite_rays_triplane(
    const uint32_t n_alive, 
    const uint32_t n_step, 
    const float T_thresh,
    int* rays_alive, 
    scalar_t* rays_t, 
    const scalar_t* __restrict__ sigmas, 
    const scalar_t* __restrict__ rgbs, 
    const scalar_t* __restrict__ deltas, 
    const scalar_t* __restrict__ ambs_aud, 
    const scalar_t* __restrict__ ambs_eye, 
    const scalar_t* __restrict__ uncertainties, 
    scalar_t* weights_sum, scalar_t* depth, scalar_t* image, scalar_t* amb_aud_sum, scalar_t* amb_eye_sum, scalar_t* uncertainty_sum
) {
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= n_alive) return;

    const int index = rays_alive[n]; // ray id
    
    // locate 
    sigmas += n * n_step;
    rgbs += n * n_step * 3;
    deltas += n * n_step * 2;
    ambs_aud += n * n_step;
    ambs_eye += n * n_step;
    uncertainties += n * n_step;

    rays_t += index;
    weights_sum += index;
    depth += index;
    image += index * 3;
    amb_aud_sum += index;
    amb_eye_sum += index;
    uncertainty_sum += index;

    scalar_t t = rays_t[0]; // current ray's t
    
    scalar_t weight_sum = weights_sum[0];
    scalar_t d = depth[0];
    scalar_t r = image[0];
    scalar_t g = image[1];
    scalar_t b = image[2];
    scalar_t a_aud = amb_aud_sum[0];
    scalar_t a_eye = amb_eye_sum[0];
    scalar_t u = uncertainty_sum[0];

    // accumulate 
    uint32_t step = 0;
    while (step < n_step) {
        
        // ray is terminated if delta == 0
        if (deltas[0] == 0) break;
        
        const scalar_t alpha = 1.0f - __expf(- sigmas[0] * deltas[0]);

        /* 
        T_0 = 1; T_i = \prod_{j=0}^{i-1} (1 - alpha_j)
        w_i = alpha_i * T_i
        --> 
        T_i = 1 - \sum_{j=0}^{i-1} w_j
        */
        const scalar_t T = 1 - weight_sum;
        const scalar_t weight = alpha * T;
        weight_sum += weight;

        t = deltas[1];
        d += weight * t;
        r += weight * rgbs[0];
        g += weight * rgbs[1];
        b += weight * rgbs[2];
        a_aud += ambs_aud[0];
        a_eye += ambs_eye[0];
        u += weight * uncertainties[0];

        //printf("[n=%d] num_steps=%d, alpha=%f, w=%f, T=%f, sum_dt=%f, d=%f\n", n, step, alpha, weight, T, sum_delta, d);

        // ray is terminated if T is too small
        // use a larger bound to further accelerate inference
        if (T < T_thresh) break;

        // locate
        sigmas++;
        rgbs += 3;
        deltas += 2;
        step++;
        ambs_aud++;
        ambs_eye++;
        uncertainties++;
    }

    //printf("[n=%d] rgb=(%f, %f, %f), d=%f\n", n, r, g, b, d);

    // rays_alive = -1 means ray is terminated early.
    if (step < n_step) {
        rays_alive[n] = -1;
    } else {
        rays_t[0] = t;
    }

    weights_sum[0] = weight_sum; // this is the thing I needed!
    depth[0] = d;
    image[0] = r;
    image[1] = g;
    image[2] = b;
    amb_aud_sum[0] = a_aud;
    amb_eye_sum[0] = a_eye;
    uncertainty_sum[0] = u;
}


void composite_rays_triplane(const uint32_t n_alive, const uint32_t n_step, const float T_thresh, at::Tensor rays_alive, at::Tensor rays_t, at::Tensor sigmas, at::Tensor rgbs, at::Tensor deltas, at::Tensor ambs_aud, at::Tensor ambs_eye, at::Tensor uncertainties, at::Tensor weights, at::Tensor depth, at::Tensor image, at::Tensor amb_aud_sum, at::Tensor amb_eye_sum, at::Tensor uncertainty_sum) {
    static constexpr uint32_t N_THREAD = 128;
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    image.scalar_type(), "composite_rays_triplane", ([&] {
        kernel_composite_rays_triplane<<<div_round_up(n_alive, N_THREAD), N_THREAD>>>(n_alive, n_step, T_thresh, rays_alive.data_ptr<int>(), rays_t.data_ptr<scalar_t>(), sigmas.data_ptr<scalar_t>(), rgbs.data_ptr<scalar_t>(), deltas.data_ptr<scalar_t>(), ambs_aud.data_ptr<scalar_t>(), ambs_eye.data_ptr<scalar_t>(), uncertainties.data_ptr<scalar_t>(), weights.data_ptr<scalar_t>(), depth.data_ptr<scalar_t>(), image.data_ptr<scalar_t>(), amb_aud_sum.data_ptr<scalar_t>(), amb_eye_sum.data_ptr<scalar_t>(), uncertainty_sum.data_ptr<scalar_t>());
    }));
}
